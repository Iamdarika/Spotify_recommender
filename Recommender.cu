#include "hip/hip_runtime.h"
#include "Recommender.h"
#ifndef DISABLE_CUDA
#include <hip/hip_runtime.h>
#include <hipblas.h>
#endif
#include <dlfcn.h>
#include <iostream>
#include <algorithm>
#include <cmath>
#include <queue>
#include <cctype>

// CUDA error checking macro (no-op in CPU-only build)
#ifndef DISABLE_CUDA
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << _FILE_ << ":" << _LINE_ << " - " \
                      << hipGetErrorString(error) << std::endl; \
            return false; \
        } \
    } while(0)
#else
#define CUDA_CHECK(call) do { (void)(call); } while(0)
#endif

#ifndef DISABLE_CUDA
#define CUBLAS_CHECK(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error at " << _FILE_ << ":" << _LINE_ << " - Status: " << status << std::endl; \
            if (status == HIPBLAS_STATUS_NOT_INITIALIZED) std::cerr << "  HIPBLAS_STATUS_NOT_INITIALIZED" << std::endl; \
            if (status == HIPBLAS_STATUS_ALLOC_FAILED) std::cerr << "  HIPBLAS_STATUS_ALLOC_FAILED" << std::endl; \
            if (status == HIPBLAS_STATUS_INVALID_VALUE) std::cerr << "  HIPBLAS_STATUS_INVALID_VALUE" << std::endl; \
            if (status == HIPBLAS_STATUS_ARCH_MISMATCH) std::cerr << "  HIPBLAS_STATUS_ARCH_MISMATCH" << std::endl; \
            return false; \
        } \
    } while(0)
#else
#define CUBLAS_CHECK(call) do { (void)(call); } while(0)
#endif

// CUDA kernel to compute norms of feature vectors
#ifndef DISABLE_CUDA
_global_ void computeNormsKernel(const float* features, float* norms, int numSongs, int featureCount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numSongs) {
        float sum = 0.0f;
        for (int i = 0; i < featureCount; ++i) {
            float val = features[idx * featureCount + i];
            sum += val * val;
        }
        norms[idx] = sqrtf(sum);
    }
}

// CUDA kernel to normalize similarity scores by norms (final step of cosine similarity)
_global_ void normalizeSimilaritiesKernel(float* similarities, const float* norms, 
                                            float queryNorm, int numSongs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numSongs) {
        float denominator = norms[idx] * queryNorm;
        if (denominator > 1e-8f) {
            similarities[idx] = similarities[idx] / denominator;
        } else {
            similarities[idx] = 0.0f;
        }
        
        // Clamp to [-1, 1] to handle floating point errors
        similarities[idx] = fminf(1.0f, fmaxf(-1.0f, similarities[idx]));
    }
}
#endif

Recommender::Recommender() 
        : initialized(false), numSongs(0), d_features(nullptr), 
            d_queryFeature(nullptr), d_similarities(nullptr), cublasHandle(nullptr),
            cublasLibHandle(nullptr), gpuEnabled(false) {
}

Recommender::~Recommender() {
    if (d_features) hipFree(d_features);
    if (d_queryFeature) hipFree(d_queryFeature);
    if (d_similarities) hipFree(d_similarities);
    #ifndef DISABLE_CUDA
    if (cublasHandle) {
        hipblasDestroy(static_cast<hipblasHandle_t>(cublasHandle));
    }
    #endif
    if (cublasLibHandle) {
        dlclose(cublasLibHandle);
    }
}

bool Recommender::initialize(const std::vector<Song>& songs) {
    std::cout << "Initializing GPU-accelerated recommender..." << std::endl;
    
    if (songs.empty()) {
        std::cerr << "Error: Empty song database" << std::endl;
        return false;
    }
    
    
    songDatabase = songs;
    numSongs = songs.size();
    
#ifdef DISABLE_CUDA
    std::cout << "[CPU-ONLY BUILD] Skipping GPU initialization." << std::endl;
    gpuEnabled = false;
#else
    // Attempt GPU initialization but allow fallback
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess || deviceCount == 0) {
        std::cerr << "[GPU Disabled] CUDA runtime not available: " << hipGetErrorString(err) << std::endl;
        std::cerr << "Falling back to CPU similarity computation." << std::endl;
        gpuEnabled = false;
    } else {
        if (hipSetDevice(0) != hipSuccess) {
            std::cerr << "[GPU Disabled] Failed to set CUDA device 0. Falling back to CPU." << std::endl;
            gpuEnabled = false;
        } else {
            // Dynamically load cuBLAS to avoid environment issues under WSL2
            const char* libNames[] = {"libcublas.so.12", "libcublas.so", nullptr};
            for (int i = 0; libNames[i] && !cublasLibHandle; ++i) {
                cublasLibHandle = dlopen(libNames[i], RTLD_LAZY | RTLD_LOCAL);
            }
            if (!cublasLibHandle) {
                std::cerr << "[GPU Disabled] Failed to load cuBLAS shared library: " << dlerror() << std::endl;
                gpuEnabled = false;
            } else {
                hipblasHandle_t handle;
                hipblasStatus_t st = hipblasCreate(&handle);
                if (st != HIPBLAS_STATUS_SUCCESS) {
                    std::cerr << "[GPU Disabled] hipblasCreate failed with status " << st << ". Falling back to CPU." << std::endl;
                    dlclose(cublasLibHandle); cublasLibHandle = nullptr;
                    gpuEnabled = false;
                } else {
                    cublasHandle = static_cast<void*>(handle);
                    gpuEnabled = true;
                }
            }
        }
    }
#endif
    
    if (gpuEnabled) {
        // Allocate GPU memory
        size_t featureMatrixSize = numSongs * FEATURE_COUNT * sizeof(float);
        if (hipMalloc(&d_features, featureMatrixSize) != hipSuccess ||
            hipMalloc(&d_queryFeature, FEATURE_COUNT * sizeof(float)) != hipSuccess ||
            hipMalloc(&d_similarities, numSongs * sizeof(float)) != hipSuccess) {
            std::cerr << "[GPU Disabled] Memory allocation on GPU failed. Falling back to CPU." << std::endl;
            gpuEnabled = false;
        } else {
            std::vector<float> featureMatrix(numSongs * FEATURE_COUNT);
            for (int i = 0; i < numSongs; ++i) {
                for (int j = 0; j < FEATURE_COUNT; ++j) {
                    featureMatrix[i * FEATURE_COUNT + j] = songs[i].features[j];
                }
            }
            if (hipMemcpy(d_features, featureMatrix.data(), featureMatrixSize, hipMemcpyHostToDevice) != hipSuccess) {
                std::cerr << "[GPU Disabled] Failed to copy feature matrix to GPU. Falling back to CPU." << std::endl;
                gpuEnabled = false;
            } else {
                std::cout << "Successfully initialized with " << numSongs << " songs on GPU" << std::endl;
            }
        }
    }
    if (!gpuEnabled) {
        std::cout << "Operating in CPU fallback mode (cosine similarity on CPU)." << std::endl;
    }
    
    initialized = true;
    return true;
}

void Recommender::calculateSimilarities(int queryIndex, float* similarities) {
    if (!initialized || queryIndex < 0 || queryIndex >= numSongs) {
        std::cerr << "Error: Invalid query index or recommender not initialized" << std::endl;
        return;
    }
    if (!gpuEnabled) {
        calculateSimilaritiesCPU(queryIndex, similarities);
        return;
    }
    
    #ifndef DISABLE_CUDA
    hipblasHandle_t handle = static_cast<hipblasHandle_t>(cublasHandle);
    #endif
    
    // Copy query feature to device
    float* queryFeatureHost = songDatabase[queryIndex].features;
    hipMemcpy(d_queryFeature, queryFeatureHost, 
               FEATURE_COUNT * sizeof(float), hipMemcpyHostToDevice);
    
    // Compute dot products using cuBLAS SGEMV
    // similarities = features * queryFeature
    // features is (numSongs x FEATURE_COUNT) matrix
    // queryFeature is (FEATURE_COUNT x 1) vector
    // result is (numSongs x 1) vector
    
    float alpha = 1.0f;
    float beta = 0.0f;
    
    // cuBLAS uses column-major, but we can treat our row-major as transposed
    // We want: d_similarities = d_features * d_queryFeature
    // In column-major interpretation: d_similarities = d_features^T * d_queryFeature
    // So we use SGEMV with transpose
    #ifndef DISABLE_CUDA
    hipblasSgemv(handle, HIPBLAS_OP_T,
                FEATURE_COUNT, numSongs,
                &alpha,
                d_features, FEATURE_COUNT,
                d_queryFeature, 1,
                &beta,
                d_similarities, 1);
    #endif
    
    // Allocate device memory for norms
    #ifndef DISABLE_CUDA
    float* d_norms;
    hipMalloc(&d_norms, numSongs * sizeof(float));
    
    // Compute norms of all feature vectors
    int blockSize = 256;
    int numBlocks = (numSongs + blockSize - 1) / blockSize;
    computeNormsKernel<<<numBlocks, blockSize>>>(d_features, d_norms, numSongs, FEATURE_COUNT);
    
    // Compute norm of query vector
    float queryNorm = 0.0f;
    for (int i = 0; i < FEATURE_COUNT; ++i) {
        queryNorm += queryFeatureHost[i] * queryFeatureHost[i];
    }
    queryNorm = std::sqrt(queryNorm);
    
    // Normalize similarities to get cosine similarity
    normalizeSimilaritiesKernel<<<numBlocks, blockSize>>>(d_similarities, d_norms, 
                                                          queryNorm, numSongs);
    
    // Copy results back to host
    hipMemcpy(similarities, d_similarities, numSongs * sizeof(float), 
               hipMemcpyDeviceToHost);
    
    // Free temporary memory
    hipFree(d_norms);
    #endif
}

void Recommender::calculateSimilaritiesCPU(int queryIndex, float* similarities) const {
    const float* query = songDatabase[queryIndex].features;
    // Pre-compute query norm
    float queryNorm = 0.0f;
    for (int j = 0; j < FEATURE_COUNT; ++j) queryNorm += query[j] * query[j];
    queryNorm = std::sqrt(queryNorm);
    for (int i = 0; i < numSongs; ++i) {
        const float* feat = songDatabase[i].features;
        float dot = 0.0f;
        float norm = 0.0f;
        for (int j = 0; j < FEATURE_COUNT; ++j) {
            dot += query[j] * feat[j];
            norm += feat[j] * feat[j];
        }
        norm = std::sqrt(norm) * queryNorm;
        similarities[i] = (norm > 1e-8f) ? std::max(-1.0f, std::min(1.0f, dot / norm)) : 0.0f;
    }
}

std::vector<int> Recommender::recommendByIndex(int songIndex, int topN) {
    if (!initialized) {
        std::cerr << "Error: Recommender not initialized" << std::endl;
        return {};
    }
    
    if (songIndex < 0 || songIndex >= numSongs) {
        std::cerr << "Error: Invalid song index: " << songIndex << std::endl;
        return {};
    }
    
    // Allocate host memory for similarities
    std::vector<float> similarities(numSongs);
    
    // Calculate similarities on GPU
    calculateSimilarities(songIndex, similarities.data());
    
    // Use a min-heap to find top-N recommendations
    std::priority_queue<Recommendation> heap;
    
    for (int i = 0; i < numSongs; ++i) {
        if (i == songIndex) continue; // Skip the query song itself
        
        Recommendation rec(i, similarities[i]);
        
        if (heap.size() < static_cast<size_t>(topN)) {
            heap.push(rec);
        } else if (rec.similarity > heap.top().similarity) {
            heap.pop();
            heap.push(rec);
        }
    }
    
    // Extract results and reverse to get descending order
    std::vector<int> results;
    results.reserve(heap.size());
    while (!heap.empty()) {
        results.push_back(heap.top().songIndex);
        heap.pop();
    }
    std::reverse(results.begin(), results.end());
    
    return results;
}

int Recommender::findSongByTrackId(const std::string& trackId) const {
    for (int i = 0; i < numSongs; ++i) {
        if (songDatabase[i].track_id == trackId) {
            return i;
        }
    }
    return -1;
}

std::string Recommender::toLower(const std::string& str) {
    std::string result = str;
    std::transform(result.begin(), result.end(), result.begin(),
                   [](unsigned char c) { return std::tolower(c); });
    return result;
}

int Recommender::findSongByName(const std::string& trackName) const {
    std::string lowerQuery = toLower(trackName);
    
    // First try exact match (case-insensitive)
    for (int i = 0; i < numSongs; ++i) {
        if (toLower(songDatabase[i].track_name) == lowerQuery) {
            return i;
        }
    }
    
    // If no exact match, try substring match
    for (int i = 0; i < numSongs; ++i) {
        if (toLower(songDatabase[i].track_name).find(lowerQuery) != std::string::npos) {
            return i;
        }
    }
    
    return -1;
}

std::vector<int> Recommender::recommend(const std::string& trackId, int topN) {
    int index = findSongByTrackId(trackId);
    if (index == -1) {
        std::cerr << "Error: Song with track_id '" << trackId << "' not found" << std::endl;
        return {};
    }
    return recommendByIndex(index, topN);
}

std::vector<int> Recommender::recommendByName(const std::string& trackName, int topN) {
    int index = findSongByName(trackName);
    if (index == -1) {
        std::cerr << "Error: Song with name '" << trackName << "' not found" << std::endl;
        return {};
    }
    return recommendByIndex(index, topN);
}

